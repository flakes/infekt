#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2010 cxxjoe
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 **/

#include "cutil_inline.h"


// how much parallelization?
#define NUM_THREADS 128
#define PER_THREAD(a_total) ((unsigned int)ceil((float)(a_total + 1) / (float)NUM_THREADS))


// largely based on
// http://mxr.mozilla.org/mozilla1.9.2/source/gfx/thebes/src/gfxBlur.cpp
// (GPL v2)


// mozilla code stuff
typedef int PRInt32;
#define PR_MAX(a, b) ((a) > (b) ? (a) : (b))
#define PR_MIN(a, b) ((a) < (b) ? (a) : (b))


__global__ void BoxBlurA8Horizontal_Device(unsigned char* aInput,
	unsigned char* aOutput,
	int aLeftLobe, int aRightLobe, int aStride, int aRows)
{
	int y = (__mul24(blockIdx.x, blockDim.x) + threadIdx.x);
	if(y >= aRows || y < 0) return;

	PRInt32 boxSize = aLeftLobe + aRightLobe + 1;
	PRInt32 baseOffset = aStride * y;

	PRInt32 alphaSum = 0;
	for (PRInt32 i = 0; i < boxSize; i++) {
		PRInt32 pos = i - aLeftLobe;
		if(pos < 0) pos = 0;
		pos = PR_MIN(pos, aStride - 1);
		alphaSum += aInput[baseOffset + pos];
	}
	for (PRInt32 x = 0; x < aStride; x++) {
		PRInt32 tmp = x - aLeftLobe;
		PRInt32 last = PR_MAX(tmp, 0);
		PRInt32 next = PR_MIN(tmp + boxSize, aStride - 1);

		aOutput[baseOffset + x] = alphaSum/boxSize;

		alphaSum += aInput[baseOffset + next] -
				aInput[baseOffset + last];
	}
}


__global__ void BoxBlurA8Vertical_Device(unsigned char* aInput,
	unsigned char* aOutput,
	int aTopLobe, int aBottomLobe, int aStride, int aRows)
{
	int x = (__mul24(blockIdx.x, blockDim.x) + threadIdx.x);
	if(x >= aStride || x < 0) return;

	PRInt32 boxSize = aTopLobe + aBottomLobe + 1;

	PRInt32 alphaSum = 0;
	for (PRInt32 i = 0; i < boxSize; i++) {
		PRInt32 pos = i - aTopLobe;
		if(pos < 0) pos = 0;
		pos = PR_MIN(pos, aRows - 1);
		alphaSum += aInput[__mul24(aStride, pos) + x];
	}
	for (PRInt32 y = 0; y < aRows; y++) {
		PRInt32 tmp = y - aTopLobe;
		PRInt32 last = PR_MAX(tmp, 0);
		PRInt32 next = PR_MIN(tmp + boxSize, aRows - 1);

		aOutput[__mul24(aStride, y) + x] = alphaSum/boxSize;

		alphaSum += aInput[__mul24(aStride, next) + x] -
				aInput[__mul24(aStride, last) + x];
	}
}


extern "C" int BoxBlurA8_Device(unsigned char* a_data, int a_stride,
	int a_rows, int a_lobes[3][2], bool a_mapped)
{
	unsigned char *l_devImgBuf, *l_devTmpBuf;
	int l_bufLen = a_stride * a_rows;

	if(!a_mapped)
	{
		hipMalloc((void**)&l_devImgBuf, l_bufLen);
		hipMemcpy(l_devImgBuf, a_data, l_bufLen, hipMemcpyHostToDevice);
	}
	else
	{
		hipHostGetDevicePointer((void**)&l_devImgBuf, (void*)a_data, 0);
	}

	hipMalloc((void**)&l_devTmpBuf, l_bufLen);

	// zomg!
    BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[0][0], a_lobes[0][1], a_stride, a_rows);
	BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[1][0], a_lobes[1][1], a_stride, a_rows);
	BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[2][0], a_lobes[2][1], a_stride, a_rows);

    BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[0][0], a_lobes[0][1], a_stride, a_rows);
	BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[1][0], a_lobes[1][1], a_stride, a_rows);
	BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[2][0], a_lobes[2][1], a_stride, a_rows);

	if(!a_mapped)
	{
		hipMemcpy(a_data, l_devImgBuf, l_bufLen, hipMemcpyDeviceToHost);
		hipFree(l_devImgBuf);
	}

	hipFree(l_devTmpBuf);

	return 1;
}
