#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2010 cxxjoe
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 **/

#include "cutil_inline.h"


// how much parallelization?
#define NUM_THREADS 128
#define PER_THREAD(a_total) ((unsigned int)ceil((float)(a_total + 1) / (float)NUM_THREADS))


// largely based on
// http://mxr.mozilla.org/mozilla1.9.2/source/gfx/thebes/src/gfxBlur.cpp
// (GPL v2)


// mozilla code stuff
typedef int PRInt32;
#define PR_MAX(a, b) ((a) > (b) ? (a) : (b))
#define PR_MIN(a, b) ((a) < (b) ? (a) : (b))


__global__ void BoxBlurA8Horizontal_Device(unsigned char* aInput,
	unsigned char* aOutput,
	int aLeftLobe, int aRightLobe, int aStride, int aRows)
{
	unsigned int y = (__umul24(blockIdx.x, blockDim.x) + threadIdx.x);
	if(y >= aRows) return;

	PRInt32 boxSize = aLeftLobe + aRightLobe + 1;

    PRInt32 alphaSum = 0;
    for (PRInt32 i = 0; i < boxSize; i++) {
        PRInt32 pos = i - aLeftLobe;
        pos = PR_MAX(pos, 0);
        pos = PR_MIN(pos, aStride - 1);
        alphaSum += aInput[aStride * y + pos];
    }
    for (PRInt32 x = 0; x < aStride; x++) {
        PRInt32 tmp = x - aLeftLobe;
        PRInt32 last = PR_MAX(tmp, 0);
        PRInt32 next = PR_MIN(tmp + boxSize, aStride - 1);

        aOutput[aStride * y + x] = alphaSum/boxSize;

        alphaSum += aInput[aStride * y + next] -
                    aInput[aStride * y + last];
    }
}


__global__ void BoxBlurA8Vertical_Device(unsigned char* aInput,
	unsigned char* aOutput,
	int aTopLobe, int aBottomLobe, int aStride, int aRows)
{
	unsigned int x = (__umul24(blockIdx.x, blockDim.x) + threadIdx.x);
	if(x >= aStride) return;

    PRInt32 boxSize = aTopLobe + aBottomLobe + 1;

    PRInt32 alphaSum = 0;
    for (PRInt32 i = 0; i < boxSize; i++) {
        PRInt32 pos = i - aTopLobe;
        pos = PR_MAX(pos, 0);
        pos = PR_MIN(pos, aRows - 1);
        alphaSum += aInput[aStride * pos + x];
    }
    for (PRInt32 y = 0; y < aRows; y++) {
        PRInt32 tmp = y - aTopLobe;
        PRInt32 last = PR_MAX(tmp, 0);
        PRInt32 next = PR_MIN(tmp + boxSize, aRows - 1);

        aOutput[aStride * y + x] = alphaSum/boxSize;

        alphaSum += aInput[aStride * next + x] -
                    aInput[aStride * last + x];
    }
}


extern "C" int BoxBlurA8_Device(unsigned char* a_data, int a_stride,
	int a_rows, int a_lobes[3][2], bool a_mapped)
{
	unsigned char *l_devImgBuf, *l_devTmpBuf;
	int l_bufLen = a_stride * a_rows;

	if(!a_mapped)
	{
		hipMalloc((void**)&l_devImgBuf, l_bufLen);
		hipMemcpy(l_devImgBuf, a_data, l_bufLen, hipMemcpyHostToDevice);
	}
	else
	{
		hipHostGetDevicePointer((void**)&l_devImgBuf, (void*)a_data, 0);
	}

	hipMalloc((void**)&l_devTmpBuf, l_bufLen);

	// zomg!
    BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[0][0], a_lobes[0][1], a_stride, a_rows);
	BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[1][0], a_lobes[1][1], a_stride, a_rows);
	BoxBlurA8Horizontal_Device<<<PER_THREAD(a_rows), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[2][0], a_lobes[2][1], a_stride, a_rows);

    BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[0][0], a_lobes[0][1], a_stride, a_rows);
	BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devImgBuf, l_devTmpBuf, a_lobes[1][0], a_lobes[1][1], a_stride, a_rows);
	BoxBlurA8Vertical_Device<<<PER_THREAD(a_stride), NUM_THREADS>>>(l_devTmpBuf, l_devImgBuf, a_lobes[2][0], a_lobes[2][1], a_stride, a_rows);

	if(!a_mapped)
	{
		hipMemcpy(a_data, l_devImgBuf, l_bufLen, hipMemcpyDeviceToHost);
		hipFree(l_devImgBuf);
	}

	hipFree(l_devTmpBuf);

	return 1;
}
